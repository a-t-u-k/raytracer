#include "hip/hip_runtime.h"
#include <iostream>
#include "accel.h"

__global__ void refraction_cuda(float *d_ray_directions_x, float *d_ray_directions_y, float *d_ray_directions_z, float *d_ray_origins_x, float *d_ray_origins_y, float *d_ray_origins_z, float *d_normals_x, float *d_normals_y, float *d_normals_z, int *d_is_propagating, float *d_t_closest, int Size, float n0, float n1)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if (index<Size)
	{
		if (d_is_propagating[index]==1)
		{
			// the ray is still propagating.

			float3 ray_direction;
			float3 plane_normal;

			ray_direction.x=d_ray_directions_x[index];
			ray_direction.y=d_ray_directions_y[index];
			ray_direction.z=d_ray_directions_z[index];
			plane_normal.x=d_normals_x[index];
			plane_normal.y=d_normals_y[index];
			plane_normal.z=d_normals_z[index];

			// normalize:
			float norm=sqrt(ray_direction.x*ray_direction.x+ray_direction.y*ray_direction.y+ray_direction.z*ray_direction.z);
			ray_direction.x=ray_direction.x/norm;
			ray_direction.y=ray_direction.y/norm;
			ray_direction.z=ray_direction.z/norm;

			norm=sqrt(plane_normal.x*plane_normal.x+plane_normal.y*plane_normal.y+plane_normal.z*plane_normal.z);
			plane_normal.x=plane_normal.x/norm;
			plane_normal.y=plane_normal.y/norm;
			plane_normal.z=plane_normal.z/norm;

			// dot product:
			float cos_incoming_alpha = plane_normal.x*ray_direction.x + plane_normal.y*ray_direction.y + plane_normal.z*ray_direction.z;
		    if (cos_incoming_alpha<0){ // normal vector is directed towards ray origin
		        cos_incoming_alpha=-cos_incoming_alpha;
		        plane_normal.x=-plane_normal.x;
		        plane_normal.y=-plane_normal.y;
		        plane_normal.z=-plane_normal.z;
		        // normal vector is directed towards the ray propagation
		    }

		    float incoming_alpha=acos(cos_incoming_alpha);
		    float sin_outcoming_alpha=n0*sin(incoming_alpha)/n1;
		    if (sin_outcoming_alpha>=1.0){
		        // terminate ray propagation - total internal reflection
		    	d_is_propagating[index]=0;
		        return;
		    }

		    // update the ray origin:
		    d_ray_origins_x[index] += d_t_closest[index]*d_ray_directions_x[index];
		    d_ray_origins_y[index] += d_t_closest[index]*d_ray_directions_y[index];
		    d_ray_origins_z[index] += d_t_closest[index]*d_ray_directions_z[index];

		    float outcoming_alpha=asin(sin_outcoming_alpha);

		    if (sin_outcoming_alpha>0){
		    	// vector orthogonal to the refraction plane
		    	float3 n_ort;
		    	n_ort.x = plane_normal.y * ray_direction.z - plane_normal.z * ray_direction.y;
		    	n_ort.y = plane_normal.z * ray_direction.x - plane_normal.x * ray_direction.z;
		    	n_ort.z = plane_normal.x * ray_direction.y - plane_normal.y * ray_direction.x;
		        // vector in the refr. plane perpend. to normal
		    	float3 n_perp;
		    	n_perp.x = plane_normal.y * n_ort.z - plane_normal.z * n_ort.y;
		    	n_perp.y = plane_normal.z * n_ort.x - plane_normal.x * n_ort.z;
		    	n_perp.z = plane_normal.x * n_ort.y - plane_normal.y * n_ort.x;

		    	// normalize just in case:
				float norm=sqrt(n_perp.x*n_perp.x+n_perp.y*n_perp.y+n_perp.z*n_perp.z);
				n_perp.x=n_perp.x/norm;
				n_perp.y=n_perp.y/norm;
				n_perp.z=n_perp.z/norm;

				float3 dir1;
				float3 dir2;
				float3 dir;

				// one potential point on the refracted ray:
				dir1.x = plane_normal.x + n_perp.x*tan(outcoming_alpha);
				dir1.y = plane_normal.y + n_perp.y*tan(outcoming_alpha);
				dir1.z = plane_normal.z + n_perp.z*tan(outcoming_alpha);

				// another potential point on the refracted ray:
				dir2.x = plane_normal.x - n_perp.x*tan(outcoming_alpha);
				dir2.y = plane_normal.y - n_perp.y*tan(outcoming_alpha);
				dir2.z = plane_normal.z - n_perp.z*tan(outcoming_alpha);

				// select proper point:
		    	float result = n_perp.x*d_ray_directions_x[index]+n_perp.y*d_ray_directions_y[index]+n_perp.z*d_ray_directions_z[index];
		    	dir=dir1;
		    	if (result<0)
		    	{
		    		// we take dir2:
		    		dir=dir2;
		    	}
		    	// normalize the direction and assign it as new direction:
		    	norm = sqrt(dir.x*dir.x+dir.y*dir.y+dir.z*dir.z);
		    	dir.x=dir.x/norm; dir.y=dir.y/norm; dir.z=dir.z/norm;

		    	d_ray_directions_x[index]=dir.x;
		    	d_ray_directions_y[index]=dir.y;
		    	d_ray_directions_z[index]=dir.z;
			}
		    //else{
		    //  // sin_outcoming_alpha==0 e.g. ray is continuing straight
			//	// its possible only if incoming_angle=0
		    //}
		}
	}
}


void callCudaFnc(vector<Ray> *rays, std::list<int> *active_rays, float n0, float n1)
{
	int Size = active_rays->size();

	int i_size = Size*sizeof(int);
	int f_size = Size*sizeof(float);

    float *ray_directions_x;
    float *ray_directions_y;
    float *ray_directions_z;
    float *ray_origins_x;
    float *ray_origins_y;
    float *ray_origins_z;
    float *normals_x;
    float *normals_y;
    float *normals_z;
    int *is_propagating;
    float *t_closest;

    ray_directions_x = (float*) malloc(f_size);
    ray_directions_y = (float*) malloc(f_size);
    ray_directions_z = (float*) malloc(f_size);

    ray_origins_x = (float*) malloc(f_size);
    ray_origins_y = (float*) malloc(f_size);
    ray_origins_z = (float*) malloc(f_size);

    normals_x = (float*) malloc(f_size);
    normals_y = (float*) malloc(f_size);
    normals_z = (float*) malloc(f_size);

    is_propagating = (int*) malloc(i_size);
    t_closest = (float*) malloc(f_size);

    int local_idx=0;
    for (std::list<int>::iterator it=active_rays->begin(); it!=active_rays->end(); ++it)
    {
    	int idx = *it;
    	// Ray:
    	ray_directions_x[local_idx]=(*rays)[idx].direction.at<float>(0);
    	ray_directions_y[local_idx]=(*rays)[idx].direction.at<float>(1);
    	ray_directions_z[local_idx]=(*rays)[idx].direction.at<float>(2);

    	ray_origins_x[local_idx]=(*rays)[idx].origin.at<float>(0);
    	ray_origins_y[local_idx]=(*rays)[idx].origin.at<float>(1);
    	ray_origins_z[local_idx]=(*rays)[idx].origin.at<float>(2);

    	normals_x[local_idx]=(*rays)[idx].normal.at<float>(0);
    	normals_y[local_idx]=(*rays)[idx].normal.at<float>(1);
    	normals_z[local_idx]=(*rays)[idx].normal.at<float>(2);

    	t_closest[local_idx]=(*rays)[idx].t_closest;
    	is_propagating[local_idx]=(*rays)[idx].is_propagating;
    	++local_idx;
    }

    float *d_ray_directions_x;
    float *d_ray_directions_y;
    float *d_ray_directions_z;
    float *d_ray_origins_x;
    float *d_ray_origins_y;
    float *d_ray_origins_z;
    float *d_normals_x;
    float *d_normals_y;
    float *d_normals_z;
    int *d_is_propagating;
    float *d_t_closest;

    hipMalloc(&d_ray_directions_x, f_size);
    hipMalloc(&d_ray_directions_y, f_size);
    hipMalloc(&d_ray_directions_z, f_size);

    hipMalloc(&d_ray_origins_x, f_size);
    hipMalloc(&d_ray_origins_y, f_size);
    hipMalloc(&d_ray_origins_z, f_size);

    hipMalloc(&d_normals_x, f_size);
    hipMalloc(&d_normals_y, f_size);
    hipMalloc(&d_normals_z, f_size);

	hipMalloc(&d_is_propagating, i_size);
	hipMalloc(&d_t_closest, f_size);

	hipMemcpy(d_ray_directions_x, ray_directions_x, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ray_directions_y, ray_directions_y, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ray_directions_z, ray_directions_z, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ray_origins_x, ray_origins_x, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ray_origins_y, ray_origins_y, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ray_origins_z, ray_origins_z, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_normals_x, normals_x, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_normals_y, normals_y, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_normals_z, normals_z, f_size, hipMemcpyHostToDevice);
	hipMemcpy(d_is_propagating, is_propagating, i_size, hipMemcpyHostToDevice);
	hipMemcpy(d_t_closest, t_closest, f_size, hipMemcpyHostToDevice);

	// refraction kernel call:
	int block_size=128;
	int grid_size=Size/block_size+1;
	refraction_cuda<<<grid_size,block_size>>>(d_ray_directions_x, d_ray_directions_y, d_ray_directions_z, d_ray_origins_x, d_ray_origins_y, d_ray_origins_z, d_normals_x, d_normals_y, d_normals_z, d_is_propagating, d_t_closest, Size, n0, n1);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();

	hipMemcpy(ray_directions_x, d_ray_directions_x, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(ray_directions_y, d_ray_directions_y, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(ray_directions_z, d_ray_directions_z, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(ray_origins_x, d_ray_origins_x, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(ray_origins_y, d_ray_origins_y, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(ray_origins_z, d_ray_origins_z, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(normals_x, d_normals_x, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(normals_y, d_normals_y, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(normals_z, d_normals_z, f_size, hipMemcpyDeviceToHost);
	hipMemcpy(is_propagating, d_is_propagating, i_size, hipMemcpyDeviceToHost);
	hipMemcpy(t_closest, d_t_closest, f_size, hipMemcpyDeviceToHost);

	// Free device memory:
	hipFree(d_ray_directions_x);
	hipFree(d_ray_directions_y);
	hipFree(d_ray_directions_z);
	hipFree(d_ray_origins_x);
	hipFree(d_ray_origins_y);
	hipFree(d_ray_origins_z);
	hipFree(d_normals_x);
	hipFree(d_normals_y);
	hipFree(d_normals_z);
	hipFree(d_is_propagating);
	hipFree(d_t_closest);

	local_idx=0;
	for (std::list<int>::iterator it=active_rays->begin(); it!=active_rays->end(); ++it)
	{
    	int idx = *it;
        // Copy the data back to ray:
    	(*rays)[idx].direction.at<float>(0)=ray_directions_x[local_idx];
    	(*rays)[idx].direction.at<float>(1)=ray_directions_y[local_idx];
    	(*rays)[idx].direction.at<float>(2)=ray_directions_z[local_idx];
    	(*rays)[idx].origin.at<float>(0)=ray_origins_x[local_idx];
    	(*rays)[idx].origin.at<float>(1)=ray_origins_y[local_idx];
    	(*rays)[idx].origin.at<float>(2)=ray_origins_z[local_idx];
    	(*rays)[idx].normal.at<float>(0)=normals_x[local_idx];
    	(*rays)[idx].normal.at<float>(1)=normals_y[local_idx];
    	(*rays)[idx].normal.at<float>(2)=normals_z[local_idx];
    	(*rays)[idx].t_closest=t_closest[local_idx];
    	(*rays)[idx].is_propagating=is_propagating[local_idx];
    	++local_idx;
    }

    free(ray_directions_x);
    free(ray_directions_y);
    free(ray_directions_z);
    free(ray_origins_x);
    free(ray_origins_y);
    free(ray_origins_z);
	free(normals_x);
	free(normals_y);
	free(normals_z);
	free(is_propagating);
	free(t_closest);
}


